#include "hip/hip_runtime.h"
/**
 * This program is a bmp image grayscale converter
 * @author Richard Zhang {zhank20@wfu.edu}
 * @date Feb.1, 2023
 * @assignment Lab 1
 * @course CSC 347
 **/

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include "bitmap_image.hpp"

using namespace std;

__global__ void color_to_grey(uchar3 *input_image, uchar3 *output_image, int width, int height)
{
    // TODO: Convert color to grayscale by mapping components of uchar3 to RGB
    // x -> R; y -> G; z -> B
    // Apply the formula:
    // output = 0.299f * R + 0.578f * G + 0.114f * B
    // Hint: First create a mapping from 2D block and grid locations to an
    // absolute 2D location in the image then use that to calculate a 1D offset
    int column = threadIdx.x + blockIdx.x * blockDim.x; 
    int row = threadIdx.y + blockIdx.y * blockDim.y; 
    if (column < width && row < height) {
        // get the linearized coordinate of the pixel we are dealing with
        int greyOffset = row * width + column;
        unsigned char r = input_image[greyOffset].x;
        unsigned char g = input_image[greyOffset].y; 
        unsigned char b = input_image[greyOffset].z;
        // do the calculation and apply it to the three components of the uchar3 respectively
        output_image[greyOffset].x = (0.299f*r + 0.578f*g + 0.114f*b);
        output_image[greyOffset].y = (0.299f*r + 0.578f*g + 0.114f*b);
        output_image[greyOffset].z = (0.299f*r + 0.578f*g + 0.114f*b);
 
    }
}


int main(int argc, char **argv)
{
    if (argc != 2) {
        cerr << "format: " << argv[0] << " { 24-bit BMP Image Filename }" << endl;
        exit(1);
    }
    
    bitmap_image bmp(argv[1]);

    if(!bmp)
    {
        cerr << "Image not found" << endl;
        exit(1);
    }

    int height = bmp.height();
    int width = bmp.width();
    
    cout << "Image dimensions:" << endl;
    cout << "height: " << height << " width: " << width << endl;

    cout << "Converting " << argv[1] << " from color to grayscale..." << endl;

    //Transform image into vector of doubles
    vector<uchar3> input_image;
    rgb_t color;
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            bmp.get_pixel(x, y, color);
            input_image.push_back( {color.red, color.green, color.blue} );
        }
    }

    vector<uchar3> output_image(input_image.size());

    uchar3 *d_in, *d_out;
    int img_size = (input_image.size() * sizeof(char) * 3);
    hipMalloc(&d_in, img_size);
    hipMalloc(&d_out, img_size);

    hipMemcpy(d_in, input_image.data(), img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input_image.data(), img_size, hipMemcpyHostToDevice);

    // TODO: Fill in the correct blockSize and gridSize
    // currently only one block with one thread is being launched
    dim3 dimGrid(ceil(width/16.0), ceil(height/16.0), 1);
    dim3 dimBlock(16, 16, 1);

    color_to_grey<<< dimGrid , dimBlock >>> (d_in, d_out, width, height);
    hipDeviceSynchronize();

    hipMemcpy(output_image.data(), d_out, img_size, hipMemcpyDeviceToHost);
    
    
    //Set updated pixels
    for(int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int pos = x * height + y;
            bmp.set_pixel(x, y, output_image[pos].x, output_image[pos].y, output_image[pos].z);
        }
    }

    cout << "Conversion complete." << endl;
    
    bmp.save_image("./grayscaledTEST_Demo.bmp");

    hipFree(d_in);
    hipFree(d_out);
}